extern "C" {
#include "CudaKernels.cuh"
}

void cudaKnn(KNNDataset *knnDataset, KNNClassifier *knnClassifier) {
	// get properties of cuda device 0
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);

	// make sure that the device has enough global memory to store the
	// entire dataset
	// TODO just make sure that the training set, the results and a
	// minimal amount of testing samples fit in the gpu global memory
	unsigned long long globalMemMinSize = ((knnDataset->numberTraining + knnDataset->numberTesting) * (knnDataset->numberFeatures + 1)) * sizeof(float);
	assert(deviceProp.totalGlobalMem > globalMemMinSize);

	// number of threads per block is half of maximum number of threads
	// per block as multiple developers suggest that leads to the best
	// performance benefits
	unsigned int threadsPerBlock = deviceProp.maxThreadsPerBlock / 2;

	// prepare gpu launching kernels
	assert(hipFree(0) == hipSuccess);

	printf("\033[1m[FATAL]:\033[0m CUDA kernels not yet implemented.\n");
	exit(-1);
}