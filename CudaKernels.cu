extern "C" {
#include "CudaKernels.cuh"
}

void cudaKnn(KNNDataset *knnDataset, KNNClassifier *knnClassifier) {
	// prepare gpu launching kernels
	assert(hipFree(0) == hipSuccess);

	printf("\033[1m[FATAL]:\033[0m CUDA kernels not yet implemented.\n");
	exit(-1);
}