#include "hip/hip_runtime.h"
extern "C" {
#include "CudaKernels.cuh"
}

/*
* CUDA-parallelized version of the kNN algorithm.
* 
* Each block is responsible for computing the distance of one or more
* testing samples to all training samples of the dataset. Each thread 
* within a block calculates the distance between one testing sample
* and one or more training samples.
* */
void cudaKnn(KNNDataset *knnDataset, KNNClassifier *knnClassifier) {
	// get properties of cuda device 0
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);

	// make sure that the device has enough global memory to store the
	// entire dataset
	// TODO just make sure that the training set, the results and a
	// minimal amount of testing samples fit in the gpu global memory
	unsigned long long globalMemMinSize = ((knnDataset->numberTraining + knnDataset->numberTesting) * knnDataset->numberFeatures) * sizeof(float) + (knnDataset->numberTraining + knnDataset->numberTesting) * sizeof(int);
	assert(deviceProp.totalGlobalMem > globalMemMinSize);

	// make sure that there is enough shared memory per block to store
	// the vector of distances
	unsigned long long minSharedMemPerBlock = knnDataset->numberTraining * sizeof(float);
	assert(deviceProp.sharedMemPerBlock > minSharedMemPerBlock);

	// number of threads per block is half of maximum number of threads
	// per block as multiple developers suggest that leads to the best
	// performance benefits
	unsigned int threadsPerBlock = deviceProp.maxThreadsPerBlock / 2;

	// number of blocks depends on the remaining available global
	// memory (two additional auxiliary vectors per block will be
	// needed) and the maximum number of blocks
	unsigned int maxNumberOfBlocks = deviceProp.maxGridSize[0];
	unsigned long long remainingGlobalMemory = deviceProp.totalGlobalMem - globalMemMinSize;
	unsigned long long additionalMemoryPerBlock = knnDataset->numberTesting * sizeof(float) + knnDataset->numberTesting * sizeof(int);
	unsigned int allowedMaxNumberOfBlocks = remainingGlobalMemory / additionalMemoryPerBlock;

	printf("Max number of blocks: %u\n", maxNumberOfBlocks);
	printf("Allowed max number of blocks: %u\n", allowedMaxNumberOfBlocks);

	// prepare gpu launching kernels
	assert(hipFree(0) == hipSuccess);

	printf("\033[1m[FATAL]:\033[0m CUDA kernels not yet implemented.\n");
	exit(-1);
}
