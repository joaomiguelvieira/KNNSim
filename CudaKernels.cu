#include "hip/hip_runtime.h"
extern "C" {
#include "CudaKernels.cuh"
}

/*
* CUDA-parallelized version of the kNN algorithm.
* 
* Each block is responsible for computing the distance of one or more
* testing samples to all training samples of the dataset. Each thread 
* within a block calculates the distance between one testing sample
* and one or more training samples.
* */
void cudaKnn(KNNDataset *knnDataset, KNNClassifier *knnClassifier) {
	// get properties of cuda device 0
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);

	// make sure that the device has enough global memory to store the
	// entire dataset
	// TODO just make sure that the training set, the results and a
	// minimal amount of testing samples fit in the gpu global memory
	// and transfer more testing samples at the rithm that the gpu
	// processes the previous ones
	unsigned long long globalMemMinSize = ((knnDataset->numberTraining + knnDataset->numberTesting) * knnDataset->numberFeatures) * sizeof(float) + (knnDataset->numberTraining + knnDataset->numberTesting) * sizeof(int);
	assert(deviceProp.totalGlobalMem > globalMemMinSize);

	// make sure that there is enough shared memory per block to store
	// the vector of distances
	unsigned long long minSharedMemPerBlock = knnDataset->numberTraining * sizeof(float);
	assert(deviceProp.sharedMemPerBlock > minSharedMemPerBlock);

	// number of threads per block is half of maximum number of threads
	// per block as multiple developers suggest that leads to the best
	// performance benefits
	unsigned int threadsPerBlock = deviceProp.maxThreadsPerBlock / 2;

	// optimal number of blocks leads to the maximum number of threads
	// per SM to be active
	unsigned int blocksPerSM = deviceProp.maxThreadsPerMultiProcessor / threadsPerBlock;
	unsigned int numberOfBlocks = deviceProp.multiProcessorCount * blocksPerSM;
	
	// the number of blocks can, however, be limited by the available
	// amount of global memory in the device
	unsigned long long remainingGlobalMemory = deviceProp.totalGlobalMem - globalMemMinSize;
	unsigned long long additionalMemoryPerBlock = knnDataset->numberTesting * sizeof(float) + knnDataset->numberTesting * sizeof(int);
	unsigned int maxNumberOfBlocks = remainingGlobalMemory / additionalMemoryPerBlock;

	if (maxNumberOfBlocks < numberOfBlocks)
		numberOfBlocks = maxNumberOfBlocks;

	// assign the calculated properties to the classifier
	knnClassifier->cudaNumberOfBlocks = numberOfBlocks;
	knnClassifier->cudaThreadsPerBlock = threadsPerBlock;

	// allocate memory in the device
	float *trainingSamplesGPU, *testingSamplesGPU;
	int *trainingClassesGPU, *testingClassesGPU;
	void *auxVectorGPU;

	// allocate operands
	assert(hipMalloc((void **) &trainingSamplesGPU, knnDataset->numberTraining * knnDataset->numberFeatures * sizeof(float)) == hipSuccess);
	assert(hipMalloc((void **) &testingSamplesGPU,  knnDataset->numberTesting  * knnDataset->numberFeatures * sizeof(float)) == hipSuccess);
	assert(hipMalloc((void **) &trainingClassesGPU, knnDataset->numberTraining                              * sizeof(int))   == hipSuccess);

	// alloocate result vector
	assert(hipMalloc((void **) &testingClassesGPU, knnDataset->numberTesting * sizeof(int)) == hipSuccess);

	// allocate auxiliary vector
	assert(hipMalloc((void **) &auxVectorGPU, knnDataset->numberTraining * numberOfBlocks * 2 * sizeof(float)) == hipSuccess);

	// copy operands to the device
	assert(hipMemcpy(trainingSamplesGPU, knnDataset->trainingSamples[0], knnDataset->numberTraining * knnDataset->numberFeatures * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
	assert(hipMemcpy(testingSamplesGPU,  knnDataset->testingSamples[0],  knnDataset->numberTesting  * knnDataset->numberFeatures * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
	assert(hipMemcpy(trainingClassesGPU, knnDataset->trainingClasses,    knnDataset->numberTraining                              * sizeof(int),   hipMemcpyHostToDevice) == hipSuccess);

	// launch cuda kernel
	cudaKnnKernel<<<numberOfBlocks, threadsPerBlock>>>(trainingSamplesGPU, trainingClassesGPU, testingSamplesGPU, testingClassesGPU, auxVectorGPU, knnDataset->numberTraining, knnDataset->numberTesting, knnDataset->numberFeatures, knnDataset->numberClasses);
	assert(hipGetLastError() == hipSuccess);

	// retrieve results back to host
	assert(hipMemcpy(knnDataset->testingClasses, testingClassesGPU, knnDataset->numberTesting * sizeof(int), hipMemcpyDeviceToHost) == hipSuccess);

	// free device memory
	assert(hipFree(auxVectorGPU) == hipSuccess);
	assert(hipFree(testingClassesGPU) == hipSuccess);
	assert(hipFree(trainingClassesGPU) == hipSuccess);
	assert(hipFree(testingSamplesGPU) == hipSuccess);
	assert(hipFree(trainingSamplesGPU) == hipSuccess);

	for (int i = 0; i < knnDataset->numberTesting; i++)
		printf("%d ", knnDataset->testingClasses[i]);

	printf("\n");

	printf("\033[1m[FATAL]:\033[0m CUDA kernels not yet fully implemented.\n");
	exit(-1);
}

__global__
void cudaKnnKernel(float *trainingSamples, int *trainingClasses, float *testingSamples, int *testingClasses, void *auxVector, int numberTraining, int numberTesting, int numberFeatures, int numberClasses) {
	// calculate the indexes of the auxiliary arrays
	float *auxDistances = ((float *) auxVector) + (blockIdx.x * 2 * numberTraining);
	int *auxIndexes = (int *) (((int *) auxVector) + ((blockIdx.x * 2 + 1) * numberTraining));

	// each block processes the testing samples whose indexes are a
	// multiple of the block index
	for (int i = blockIdx.x; i < numberTesting; i += gridDim.x) {
		// each thread processes the training samples whose indexes
		// are a multiple of the thread index
		for (int j = threadIdx.x; j < numberTraining; j += blockDim.x) {
			// calculate distance and initialize distance index array
			auxDistances[j] = 0;
			auxIndexes[j] = j;
		}

		// sync threads
		__syncthreads();

		// thread 0 double sorts distance and index arrays

		// thread 0 does class assignement
		testingClasses[i] = -2;
	}
}