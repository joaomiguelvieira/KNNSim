#include "hip/hip_runtime.h"
extern "C" {
#include "CudaKernels.cuh"
}

/*
 * CUDA-parallelized version of the kNN algorithm (1).
 *
 * Each block is responsible for computing the distance of one or more
 * testing samples to all training samples of the dataset. Each thread
 * within a block calculates the distance between one testing sample
 * and one or more training samples.
 * */
void cudaKnn1(KNNDataset *knnDataset, KNNClassifier *knnClassifier) {
	// get properties of cuda device 0
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);

	// make sure that the device has enough global memory to store the
	// entire dataset
	unsigned long long globalMemMinSize = ((knnDataset->numberTraining + knnDataset->numberTesting) * knnDataset->numberFeatures) * sizeof(float) + (knnDataset->numberTraining + knnDataset->numberTesting) * sizeof(int);
	assert(deviceProp.totalGlobalMem > globalMemMinSize);

	// number of threads per block is half of maximum number of threads
	// per block as multiple developers suggest that leads to the best
	// performance benefits
	unsigned int threadsPerBlock = deviceProp.maxThreadsPerBlock / 2;

	// optimal number of blocks leads to the maximum number of threads
	// per SM to be active
	unsigned int blocksPerSM = deviceProp.maxThreadsPerMultiProcessor / threadsPerBlock;
	unsigned int numberOfBlocks = deviceProp.multiProcessorCount * blocksPerSM;
	
	// the number of blocks can, however, be limited by the available
	// amount of global memory in the device
	unsigned long long remainingGlobalMemory = deviceProp.totalGlobalMem - globalMemMinSize;
	unsigned long long additionalMemoryPerBlock = knnDataset->numberTraining * sizeof(float) + knnDataset->numberTraining * sizeof(int);
	unsigned int maxNumberOfBlocks = remainingGlobalMemory / additionalMemoryPerBlock;

	if (maxNumberOfBlocks < numberOfBlocks)
		numberOfBlocks = maxNumberOfBlocks;

	// make sure that there is enough remaining global memory to launch
	// at least one block
	assert(numberOfBlocks > 0);

	// assign the calculated properties to the classifier
	strcpy(knnClassifier->cudaDeviceName, deviceProp.name);
	knnClassifier->cudaPeakGlobalMemory = globalMemMinSize + numberOfBlocks * additionalMemoryPerBlock;
	knnClassifier->cudaNumberOfBlocks = numberOfBlocks;
	knnClassifier->cudaThreadsPerBlock = threadsPerBlock;
	knnClassifier->cudaDeviceUtilization = numberOfBlocks * threadsPerBlock / (deviceProp.maxThreadsPerMultiProcessor * deviceProp.multiProcessorCount);

	// allocate memory in the device
	float *trainingSamplesGPU, *testingSamplesGPU;
	int *trainingClassesGPU, *testingClassesGPU;
	void *auxVectorGPU;

	// allocate operands
	assert(hipMalloc((void **) &trainingSamplesGPU, knnDataset->numberTraining * knnDataset->numberFeatures * sizeof(float)) == hipSuccess);
	assert(hipMalloc((void **) &testingSamplesGPU,  knnDataset->numberTesting  * knnDataset->numberFeatures * sizeof(float)) == hipSuccess);
	assert(hipMalloc((void **) &trainingClassesGPU, knnDataset->numberTraining                              * sizeof(int))   == hipSuccess);

	// allocate result vector
	assert(hipMalloc((void **) &testingClassesGPU, knnDataset->numberTesting * sizeof(int)) == hipSuccess);

	// allocate auxiliary vector
	assert(hipMalloc((void **) &auxVectorGPU, knnDataset->numberTraining * numberOfBlocks * 2 * sizeof(float)) == hipSuccess);

	// copy operands to the device
	assert(hipMemcpy(trainingSamplesGPU, knnDataset->trainingSamples[0], knnDataset->numberTraining * knnDataset->numberFeatures * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
	assert(hipMemcpy(testingSamplesGPU,  knnDataset->testingSamples[0],  knnDataset->numberTesting  * knnDataset->numberFeatures * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
	assert(hipMemcpy(trainingClassesGPU, knnDataset->trainingClasses,    knnDataset->numberTraining                              * sizeof(int),   hipMemcpyHostToDevice) == hipSuccess);

	// measure cuda kernel time
	hipEvent_t cudaKernelStart, cudaKernelStop;
	assert(hipEventCreate(&cudaKernelStart) == hipSuccess);
	assert(hipEventCreate(&cudaKernelStop) == hipSuccess);

	// launch cuda kernel
	assert(hipEventRecord(cudaKernelStart) == hipSuccess);
	cudaKnnKernel1<<<numberOfBlocks, threadsPerBlock>>>(trainingSamplesGPU, trainingClassesGPU, testingSamplesGPU, testingClassesGPU, auxVectorGPU, knnDataset->numberTraining, knnDataset->numberTesting, knnDataset->numberFeatures, knnDataset->numberClasses, knnClassifier->k);
	assert(hipEventRecord(cudaKernelStop) == hipSuccess);

	// assign cuda kernel time to the classifier
	assert(hipEventSynchronize(cudaKernelStop) == hipSuccess);
	float cudaElapsedMs;
	assert(hipEventElapsedTime(&cudaElapsedMs, cudaKernelStart, cudaKernelStop) == hipSuccess);
	knnClassifier->cudaKernelTime = cudaElapsedMs / 1000;

	// check if any errors have occured
	assert(hipGetLastError() == hipSuccess);

	// retrieve results back to host
	assert(hipMemcpy(knnDataset->testingClasses, testingClassesGPU, knnDataset->numberTesting * sizeof(int), hipMemcpyDeviceToHost) == hipSuccess);

	// cleanup
	assert(hipFree(auxVectorGPU) == hipSuccess);
	assert(hipFree(testingClassesGPU) == hipSuccess);
	assert(hipFree(trainingClassesGPU) == hipSuccess);
	assert(hipFree(testingSamplesGPU) == hipSuccess);
	assert(hipFree(trainingSamplesGPU) == hipSuccess);
}

__global__
void cudaKnnKernel1(float *trainingSamples, int *trainingClasses, float *testingSamples, int *testingClasses, void *auxVector, int numberTraining, int numberTesting, int numberFeatures, int numberClasses, int k) {
	// calculate the indexes of the auxiliary arrays
	float *auxDistances = ((float *) auxVector) + (blockIdx.x * 2 * numberTraining);
	int *auxIndexes = (int *) (((int *) auxVector) + ((blockIdx.x * 2 + 1) * numberTraining));

	// each block processes the testing samples whose indexes are a
	// multiple of the block index
	for (int i = blockIdx.x; i < numberTesting; i += gridDim.x) {
		// calculate address of testing sample
		float *testingSample = testingSamples + i * numberFeatures;

		// each thread processes the training samples whose indexes
		// are a multiple of the thread index
		for (int j = threadIdx.x; j < numberTraining; j += blockDim.x) {
			// calculate distance of training sample
			float *trainingSample = trainingSamples + j * numberFeatures;

			// calculate distance and initialize distance index array
			auxDistances[j] = sumOfSquaredDifferencesGPU(testingSample, trainingSample, numberFeatures);
			auxIndexes[j] = j;
		}

		// sync threads
		__syncthreads();

		// last two phases of knn are sequential
		if (threadIdx.x == 0) {
			// thread 0 double sorts distance and index arrays
			doubleSortGPU(auxDistances, auxIndexes, numberTraining, k);

			// thread 0 does class assignement
			testingClasses[i] = findClassGPU(trainingClasses, numberClasses, k, auxIndexes, (int *) auxDistances);
		}

		// sync threads
		__syncthreads();
	}
}

__device__
float sumOfSquaredDifferencesGPU(float *sample1, float *sample2, int numberFeatures) {
  float difference, distance = 0;

  for (int i = 0; i < numberFeatures; i++) {
    difference = sample1[i] - sample2[i];
    distance += difference * difference;
  }

  return distance;
}

__device__
void doubleSortGPU(float *distances, int *indexes, int numberTraining, int k) {
  int minimum, aux;

  // sort the minimum k elements
  for (int i = 0; i < k; i++) {
    minimum = i;

    // find the next minimum value
    for (int j = i + 1; j < numberTraining; j++)
      if (distances[j] < distances[minimum])
        minimum = j;

    // exchange values
    aux = distances[i];
    distances[i] = distances[minimum];
    distances[minimum] = aux;

    // exchange indexes
    aux = indexes[i];
    indexes[i] = indexes[minimum];
    indexes[minimum] = aux;
  }
}

__device__
int findClassGPU(int *trainingClasses, int numberClasses, int k, int *indexes, int *classes) {
  int maximum = 0;

  // initialize vote array
  for (int i = 0; i < numberClasses; i++)
    classes[i] = 0;

  // voting process
  for (int i = 0; i < k; i++)
    classes[trainingClasses[indexes[i]]]++;

  // find the most voted class
  for (int i = 1; i < numberClasses; i++)
    if (classes[i] > classes[maximum])
      maximum = i;

  return maximum;
}

/*
 * CUDA parallelized version of the kNN algorithm (2).
 *
 * Each block is responsible for computing the distance of one or more
 * testing samples to all training samples of the dataset. Each thread
 * within a block calculates the distance between one testing sample
 * and one or more training samples. Then, it updates its own list of
 * the k closest samples of the training set and keeps it on a shared
 * vector. Finally, the thread with the index 0 sorts the vector and
 * determines the closest k samples.
 * */
void cudaKnn2(KNNDataset *knnDataset, KNNClassifier *knnClassifier) {
    // get properties of cuda device 0
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    // make sure that the device has enough global memory to store the
    // entire dataset
    unsigned long long globalMemMinSize = ((knnDataset->numberTraining + knnDataset->numberTesting) * knnDataset->numberFeatures) * sizeof(float) + (knnDataset->numberTraining + knnDataset->numberTesting) * sizeof(int);
    assert(deviceProp.totalGlobalMem > globalMemMinSize);

    // number of threads per block is half of maximum number of threads
    // per block as multiple developers suggest that leads to the best
    // performance benefits
    unsigned int threadsPerBlock = deviceProp.maxThreadsPerBlock / 2;

    // optimal number of blocks leads to the maximum number of threads
    // per SM to be active
    unsigned int blocksPerSM = deviceProp.maxThreadsPerMultiProcessor / threadsPerBlock;
    unsigned int numberOfBlocks = deviceProp.multiProcessorCount * blocksPerSM;

    // the number of blocks can, however, be limited by the available
    // amount of shared memory in the device
    unsigned long long requiredSharedMemoryPerThread = knnClassifier->k * (sizeof(float) + sizeof(int));
    unsigned long long requiredSharedMemoryPerBlock = requiredSharedMemoryPerThread * threadsPerBlock;
    assert(requiredSharedMemoryPerBlock <= deviceProp.sharedMemPerBlock);

    // assign the calculated properties to the classifier
    strcpy(knnClassifier->cudaDeviceName, deviceProp.name);
    knnClassifier->cudaPeakGlobalMemory = globalMemMinSize;
    knnClassifier->cudaNumberOfBlocks = numberOfBlocks;
    knnClassifier->cudaThreadsPerBlock = threadsPerBlock;
    knnClassifier->cudaDeviceUtilization = numberOfBlocks * threadsPerBlock / (deviceProp.maxThreadsPerMultiProcessor * deviceProp.multiProcessorCount);

    // allocate memory in the device
    float *trainingSamplesGPU, *testingSamplesGPU;
    int *trainingClassesGPU, *testingClassesGPU;

    // allocate operands
    assert(hipMalloc((void **) &trainingSamplesGPU, knnDataset->numberTraining * knnDataset->numberFeatures * sizeof(float)) == hipSuccess);
    assert(hipMalloc((void **) &testingSamplesGPU,  knnDataset->numberTesting  * knnDataset->numberFeatures * sizeof(float)) == hipSuccess);
    assert(hipMalloc((void **) &trainingClassesGPU, knnDataset->numberTraining                              * sizeof(int))   == hipSuccess);

    // allocate result vector
    assert(hipMalloc((void **) &testingClassesGPU, knnDataset->numberTesting * sizeof(int)) == hipSuccess);

    // copy operands to the device
    assert(hipMemcpy(trainingSamplesGPU, knnDataset->trainingSamples[0], knnDataset->numberTraining * knnDataset->numberFeatures * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
    assert(hipMemcpy(testingSamplesGPU,  knnDataset->testingSamples[0],  knnDataset->numberTesting  * knnDataset->numberFeatures * sizeof(float), hipMemcpyHostToDevice) == hipSuccess);
    assert(hipMemcpy(trainingClassesGPU, knnDataset->trainingClasses,    knnDataset->numberTraining                              * sizeof(int),   hipMemcpyHostToDevice) == hipSuccess);

    // measure cuda kernel time
    hipEvent_t cudaKernelStart, cudaKernelStop;
    assert(hipEventCreate(&cudaKernelStart) == hipSuccess);
    assert(hipEventCreate(&cudaKernelStop) == hipSuccess);

    exit(-1);

    // launch cuda kernel
    assert(hipEventRecord(cudaKernelStart) == hipSuccess);
    cudaKnnKernel2<<<numberOfBlocks, threadsPerBlock>>>(trainingSamplesGPU, trainingClassesGPU, testingSamplesGPU, testingClassesGPU, knnDataset->numberTraining, knnDataset->numberTesting, knnDataset->numberFeatures, knnDataset->numberClasses, knnClassifier->k);
    assert(hipEventRecord(cudaKernelStop) == hipSuccess);

    // assign cuda kernel time to the classifier
    assert(hipEventSynchronize(cudaKernelStop) == hipSuccess);
    float cudaElapsedMs;
    assert(hipEventElapsedTime(&cudaElapsedMs, cudaKernelStart, cudaKernelStop) == hipSuccess);
    knnClassifier->cudaKernelTime = cudaElapsedMs / 1000;

    // check if any errors have occurred
    assert(hipGetLastError() == hipSuccess);

    // retrieve results back to host
    assert(hipMemcpy(knnDataset->testingClasses, testingClassesGPU, knnDataset->numberTesting * sizeof(int), hipMemcpyDeviceToHost) == hipSuccess);

    // cleanup
    assert(hipFree(testingClassesGPU) == hipSuccess);
    assert(hipFree(trainingClassesGPU) == hipSuccess);
    assert(hipFree(testingSamplesGPU) == hipSuccess);
    assert(hipFree(trainingSamplesGPU) == hipSuccess);
}

__global__
void cudaKnnKernel2(float *trainingSamples, int *trainingClasses, float *testingSamples, int *testingClasses, int numberTraining, int numberTesting, int numberFeatures, int numberClasses, int k) {

}