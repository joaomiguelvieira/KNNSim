extern "C" {
#include "CudaKernels.cuh"
}

/*
* CUDA-parallelized version of the kNN algorithm.
* 
* Each block is responsible for computing the distance of one or more
* testing samples to all training samples of the dataset. Each thread 
* within a block calculates the distance between one testing sample
* and one or more training samples.
* */
void cudaKnn(KNNDataset *knnDataset, KNNClassifier *knnClassifier) {
	// get properties of cuda device 0
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);

	// make sure that the device has enough global memory to store the
	// entire dataset
	// TODO just make sure that the training set, the results and a
	// minimal amount of testing samples fit in the gpu global memory
	unsigned long long globalMemMinSize = ((knnDataset->numberTraining + knnDataset->numberTesting) * (knnDataset->numberFeatures + 1)) * sizeof(float);
	assert(deviceProp.totalGlobalMem > globalMemMinSize);

	// make sure that there is enough shared memory per block to store
	// the vector of distances
	unsigned long long minSharedMemPerBlock = knnDataset->numberTraining * sizeof(float);
	assert(deviceProp.sharedMemPerBlock > minSharedMemPerBlock);

	// number of threads per block is half of maximum number of threads
	// per block as multiple developers suggest that leads to the best
	// performance benefits
	unsigned int threadsPerBlock = deviceProp.maxThreadsPerBlock / 2;

	// number of blocks is set to 

	// prepare gpu launching kernels
	assert(hipFree(0) == hipSuccess);

	printf("\033[1m[FATAL]:\033[0m CUDA kernels not yet implemented.\n");
	exit(-1);
}